#include "hip/hip_runtime.h"
#include "virgo.cuh"
#include <iostream>
#include <string>
#include <sstream>

namespace virgo {
  // This is arkwork inverse R, not icicle inverse R.
  // inv_r = 9915499612839321149637521777990102151350674507940716049588462388200839649614
  // inv_r2 = inv_r ^ 2 = 8519677608991584271437967308266649112183478179623991153221810821821888926024
  template <typename S>
  __device__ constexpr S inv_r_mont = S({0x6db1194e, 0xdc5ba005, 0xe111ec87, 0x90ef5a9, 0xaeb85d5d, 0xc8260de4, 0x82c5551c, 0x15ebf951});

  template <typename S>
  __device__ constexpr S inv_r_mont2 = S({0xd3c71148, 0xae12ba81, 0xb38e2428, 0x52f28270, 0x79a1edeb, 0xe065f3e3, 0xe436631e, 0x12d5f775});

  /////////////////////////////////
  /// COMMON FUNCTIONS
  /////////////////////////////////

  template <typename S>
  __global__ void mul_pair_kernel(S* arr1, S* arr2, S* result, int n) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    result[tid] = arr1[tid] * arr2[tid] * (inv_r_mont<S>);
  }

  template <typename S>
  __global__ void reduce_sum_kernel(S* result, uint32_t n, uint32_t half) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto other = tid + half;
    if (other < n) {
      result[tid] = result[tid] + result[other];
    }
  }

    template <typename S>
  __global__ void reduce_sum_kernel2(S* result, uint32_t m, uint32_t n, uint32_t half, uint32_t offset) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    uint32_t arr_id = tid % m;
    uint32_t arr_pos = tid / m;

    auto other = arr_pos + half;
    if (other < n) {
      auto start = offset * arr_id + arr_pos;
      result[start] = result[start] + result[start + half];
    }
  }

  template <typename S>
  hipError_t sum_arrays(S* arrs, uint32_t m, uint32_t n)
  {
    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);
    hipStream_t stream;
    hipStreamCreate(&stream);

    auto x = n;
    while (x > 1) {
      int worker_count = (x * m + 1) >> 1;
      int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
      int num_blocks = (worker_count + num_threads - 1) / num_threads;

      int half = (x + 1) >> 1;
      reduce_sum_kernel2 <<< num_blocks, num_threads, 0, stream >>> (arrs, m, x, half, n);

      x = (x + 1) >> 1;
    }
  }

  template <typename S>
  hipError_t sum_single_array(S* arr, int n)
  {
    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);
    hipStream_t stream;
    hipStreamCreate(&stream);

    auto x = n;
    while (x > 1) {
      int worker_count = x >> 1;
      int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
      int num_blocks = (worker_count + num_threads - 1) / num_threads;

      int half = (x + 1) >> 1;
      reduce_sum_kernel <<< num_blocks, num_threads, 0, stream >>> (arr, x, half);

      x = (x + 1) >> 1;
    }
  }

  /////////////////////////////////
  /// BookKeeping sum_all
  /////////////////////////////////

  template <typename S>
  void print_arr(S* arr, int start, int end) {
    int len = end - start;
    S* tmp = (S*)malloc(len * sizeof(S));

    hipMemcpy(tmp, arr + start, len * sizeof(S), hipMemcpyDeviceToHost);
    for (int i = 0; i < len; i++) {
      std::cout << tmp[i] * inv_r_mont<S> << " ";
    }
    std::cout << std::endl;

    delete [] tmp;
  }

  template <typename S>
  hipError_t bk_sum_all_case_1(
    S* table1, S* table2, S* output, int n)
  {
    CHK_INIT_IF_RETURN();

    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);

    S* device_tmp;
    // allocate device array
    hipMalloc((void**)&device_tmp, n * sizeof(S));

    // 1. Multiply all pair values.
    int worker_count = n;
    int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
    int num_blocks = (worker_count + num_threads - 1) / num_threads;

    mul_pair_kernel <<< num_blocks, num_threads >>> (table1, table2, device_tmp, n);

    // 2. Sum up all the values in the array.
    sum_single_array(device_tmp, n);
    hipMemcpy(output, device_tmp, sizeof(S), hipMemcpyHostToHost);

    hipFree(device_tmp);

    return CHK_LAST();
  }

  template <typename S>
  hipError_t bk_sum_all_case_2(
    S* arr, S* output, int n)
  {
    CHK_INIT_IF_RETURN();

    // Sum up all the values in the array.
    sum_single_array(arr, n);
    hipMemcpy(output, arr, sizeof(S), hipMemcpyHostToHost);

    return CHK_LAST();
  }

  /////////////////////////////////
  /// BookKeeping produce
  /////////////////////////////////

  // template <typename S>
  // hipError_t bk_produce_case_1(
  //   S* table1, S* table2, S* output, int n)
  // {
  //   CHK_INIT_IF_RETURN();

  //   int cuda_device_ix = 0;
  //   hipDeviceProp_t prop;
  //   hipGetDeviceProperties(&prop, cuda_device_ix);

  //   S* device_tmp;
  //   // allocate device array
  //   hipMalloc((void**)&device_tmp, n * sizeof(S));

  //   // 1. Multiply all pair values.
  //   int worker_count = n;
  //   int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
  //   int num_blocks = (worker_count + num_threads - 1) / num_threads;

  //   mul_pair_kernel <<< num_blocks, num_threads >>> (arr1, arr2, device_tmp, n);

  //   // 2. Sum up all the values in the array.
  //   sum_single_array(device_tmp, n);
  //   hipMemcpy(output, device_tmp, sizeof(S), hipMemcpyHostToHost);

  //   hipFree(device_tmp);

  //   return CHK_LAST();
  // }

}
