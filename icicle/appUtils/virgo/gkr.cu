#include "hip/hip_runtime.h"
#include "circuit.cuh"
#include "common.cuh"

namespace virgo {
  template <typename S>
  __global__ void precompute_bookeeping_kernel(S* g, uint8_t g_index, S* prev_output, S* output)
  {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    uint32_t prev_output_size = 1 << g_index;
    uint32_t curr_output_size = prev_output_size * 2;

    if (tid >= curr_output_size) { return; };

    uint32_t prev_tid = tid >> 1;

    output[tid] = prev_output[prev_tid] * g[g_index] * inv_r_mont<S>;

    if (tid % 2 == 0) { output[tid] = prev_output[prev_tid] - output[tid]; }
  }

  template <typename S>
  hipError_t precompute_bookeeping(S init, S* g, uint8_t g_size, S* output)
  {
    CHK_INIT_IF_RETURN();

    if (g_size > 0) {
      S* intermediate_output;

      CHK_IF_RETURN(hipMalloc((void**)&intermediate_output, (1 << g_size) * sizeof(S)));
      CHK_IF_RETURN(hipMemcpy(intermediate_output, &init, sizeof(S), hipMemcpyHostToDevice));

      for (uint8_t i = 0; i < g_size; i++) {
        uint32_t curr_output_size = 1 << (i + 1);
        auto [num_blocks, num_threads] = find_thread_block(curr_output_size);
        precompute_bookeeping_kernel<<<num_blocks, num_threads>>>(g, i, intermediate_output, output);

        CHK_IF_RETURN(hipMemcpy(intermediate_output, output, curr_output_size * sizeof(S), hipMemcpyDeviceToDevice));
      }

      CHK_IF_RETURN(hipFree(intermediate_output));
    } else {
      CHK_IF_RETURN(hipMemcpy(output, &init, sizeof(S), hipMemcpyHostToDevice));
    }

    return CHK_LAST();
  }

  template <typename S>
  __global__ void update_bookeeping_phase_1(
    uint8_t num_layers, SparseMultilinearExtension<S>* f_extensions, S** s_evaluations, S* bookeeping_g, S* output)
  {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    // exts are extensions from layer i to j. So x_num_vars is the same at all
    // extensions.
    uint32_t x_num_vars = f_extensions[0].x_num_vars;

    uint32_t x_index = tid;
    uint32_t replica_index = x_index / (1 << x_num_vars);
    uint32_t relative_x_index = x_index % (1 << x_num_vars);

    for (uint8_t relative_layer_index = 0; relative_layer_index < num_layers; relative_layer_index++) {
      SparseMultilinearExtension<S> target_ext = f_extensions[relative_layer_index];

      uint32_t start = target_ext.x_indices_start[relative_x_index];
      uint32_t end = target_ext.x_indices_start[relative_x_index + 1];

      for (uint32_t position_index = start; position_index < end; position_index++) {
        uint32_t position = target_ext.x_indices[position_index];

        if (target_ext.point_x[position] != relative_x_index) { panic(); }

        uint32_t z_index = target_ext.point_z[position] + (replica_index << target_ext.z_num_vars);
        uint32_t y_index = target_ext.point_y[position] + (replica_index << target_ext.y_num_vars);

        S evaluation = target_ext.evaluations[position];

        output[x_index] = output[x_index] + bookeeping_g[z_index] * s_evaluations[relative_layer_index][y_index] *
                                              evaluation * inv_r_mont2<S>;
      }
    }
  }

  template <typename S>
  hipError_t initialize_phase_1_plus(
    uint32_t num_replicas,
    uint32_t num_layers,
    uint32_t output_size,
    SparseMultilinearExtension<S>* f_extensions,
    S** s_evaluations,
    S* bookeeping_g,
    S* output)
  {
    CHK_INIT_IF_RETURN();

    auto [num_blocks, num_threads] = find_thread_block(output_size);
    update_bookeeping_phase_1<<<num_blocks, num_threads>>>(
      num_layers, f_extensions, s_evaluations, bookeeping_g, output);

    return CHK_LAST();
  }

  template <typename S>
  __global__ void update_bookeeping_phase_2(
    uint8_t relative_layer_index,
    SparseMultilinearExtension<S>* f_extensions,
    S* bookeeping_g,
    S* bookeeping_u,
    S** output)
  {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    SparseMultilinearExtension<S> target_ext = f_extensions[relative_layer_index];

    uint32_t y_index = tid;
    uint32_t replica_index = y_index / (1 << target_ext.y_num_vars);
    uint32_t relative_y_index = y_index % (1 << target_ext.y_num_vars);

    uint32_t start = target_ext.y_indices_start[relative_y_index];
    uint32_t end = target_ext.y_indices_start[relative_y_index + 1];

    for (uint32_t position_index = start; position_index < end; position_index++) {
      uint32_t position = target_ext.y_indices[position_index];

      if (target_ext.point_y[position] != relative_y_index) { panic(); }

      uint32_t z_index = target_ext.point_z[position] + (replica_index << target_ext.z_num_vars);
      uint32_t x_index = target_ext.point_x[position] + (replica_index << target_ext.x_num_vars);

      S evaluation = target_ext.evaluations[position];

      output[relative_layer_index][y_index] = output[relative_layer_index][y_index] + bookeeping_g[z_index] *
                                                                                        bookeeping_u[x_index] *
                                                                                        evaluation * inv_r_mont2<S>;
    }
  }

  template <typename S>
  hipError_t initialize_phase_2_plus(
    uint32_t num_replicas,
    uint32_t num_layers,
    uint32_t* on_host_output_size,
    SparseMultilinearExtension<S>* f_extensions,
    S* bookeeping_g,
    S* bookeeping_u,
    S** output)
  {
    CHK_INIT_IF_RETURN();

    for (uint8_t relative_layer_index = 0; relative_layer_index < num_layers; relative_layer_index++) {
      auto [num_blocks, num_threads] = find_thread_block(on_host_output_size[relative_layer_index]);

      update_bookeeping_phase_2<<<num_blocks, num_threads>>>(
        relative_layer_index, f_extensions, bookeeping_g, bookeeping_u, output);
    }

    return CHK_LAST();
  }
} // namespace virgo
