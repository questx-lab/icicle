#include "hip/hip_runtime.h"
#include "circuit.cuh"
#include "common.cuh"

namespace virgo {
  __device__ const uint8_t GATE_CONST = 0;
  __device__ const uint8_t GATE_MULTIPLICATION = 1;
  __device__ const uint8_t GATE_FORWARD_X = 2;
  __device__ const uint8_t GATE_FORWARD_Y = 3;

  template <typename S>
  __device__ S calculate_gate(uint8_t gate_type, S c, S in1, S in2)
  {
    if (gate_type == GATE_CONST) {
      return c;
    } else if (gate_type == GATE_MULTIPLICATION) {
      S result = c * in1 * in2;
      return result * inv_r_mont2<S>;
    } else if (gate_type == GATE_FORWARD_X) {
      S result = c * in1;
      return result * inv_r_mont<S>;
    } else if (gate_type == GATE_FORWARD_Y) {
      S result = c * in2;
      return result * inv_r_mont<S>;
    } else {
      panic();
    }
  }

  template <typename S>
  __global__ void evaluate_single_gate_type(
    uint8_t num_layers,
    uint8_t layer_index,
    uint32_t size,
    uint8_t gate_type,
    SparseMultilinearExtension<S>* ext,
    ReverseSparseMultilinearExtension** reverse_exts,
    S** evaluations)
  {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto z_index = tid;

    if (z_index >= size) { return; }

    for (uint8_t target_layer_index = layer_index + 1; target_layer_index < num_layers + 1; target_layer_index++) {
      uint8_t ext_index = target_layer_index - layer_index - 1;
      SparseMultilinearExtension<S> target_ext = ext[ext_index];

      for (uint8_t i = 0; i < target_ext.z_indices_size[z_index]; i++) {
        uint32_t k = target_ext.z_indices[z_index][i];

        if (target_ext.point_z[k] != z_index) { panic(); }

        uint32_t x_index = target_ext.point_x[k];

        // Currently, y_index is a subset index, we must convert it to real index.
        uint32_t y_subset_index = target_ext.point_y[k];
        uint32_t y_index_position = reverse_exts[target_layer_index][layer_index].subset_position[y_subset_index];
        uint32_t y_real_index = reverse_exts[target_layer_index][layer_index].point_real[y_index_position];

        S c = target_ext.evaluations[k];
        S x = evaluations[layer_index + 1][x_index];
        S y = evaluations[target_layer_index][y_real_index];

        evaluations[layer_index][z_index] = evaluations[layer_index][z_index] + calculate_gate(gate_type, c, x, y);
      }
    }
  }

  template <typename S>
  hipError_t layer_evaluate(
    uint8_t num_layers,
    uint8_t layer_index,
    const Layer<S>& layer,
    ReverseSparseMultilinearExtension** reverse_exts,
    S** evaluations)
  {
    CHK_INIT_IF_RETURN();

    // We need to evaluate 2^num_vars gates of z.
    auto [num_blocks, num_threads] = find_thread_block(layer.size);

    evaluate_single_gate_type<<<num_blocks, num_threads>>>(
      num_layers, layer_index, layer.size, GATE_CONST, layer.constant_ext, reverse_exts, evaluations);

    evaluate_single_gate_type<<<num_blocks, num_threads>>>(
      num_layers, layer_index, layer.size, GATE_MULTIPLICATION, layer.mul_ext, reverse_exts, evaluations);

    evaluate_single_gate_type<<<num_blocks, num_threads>>>(
      num_layers, layer_index, layer.size, GATE_FORWARD_X, layer.forward_x_ext, reverse_exts, evaluations);

    evaluate_single_gate_type<<<num_blocks, num_threads>>>(
      num_layers, layer_index, layer.size, GATE_FORWARD_Y, layer.forward_y_ext, reverse_exts, evaluations);

    return CHK_LAST();
  }

  template <typename S>
  hipError_t circuit_evaluate(const Circuit<S>& circuit, S** evaluations)
  {
    for (int8_t layer_index = circuit.num_layers - 1; layer_index >= 0; layer_index--) {
      layer_evaluate(circuit.num_layers, layer_index, circuit.layers[layer_index], circuit.reverse_exts, evaluations);
    }

    return CHK_LAST();
  }

  template <typename S>
  __global__ void extract_subset_evaluation(
    ReverseSparseMultilinearExtension** reverse_exts,
    uint8_t source_layer_index,
    uint8_t target_layer_index,
    S** evaluations,
    S** subset_evaluations)
  {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;
    uint32_t subset_index = tid;

    ReverseSparseMultilinearExtension reverse_ext = reverse_exts[target_layer_index][source_layer_index];

    // This tid
    uint32_t point_position = reverse_ext.subset_position[subset_index];
    if (point_position == 4294967295) { return; }

    uint32_t real_index = reverse_ext.point_real[point_position];
    subset_evaluations[target_layer_index - source_layer_index - 1][subset_index] =
      evaluations[target_layer_index][real_index];
  }

  template <typename S>
  hipError_t
  circuit_subset_evaluations(const Circuit<S>& circuit, uint8_t layer_index, S** evaluations, S** subset_evaluations)
  {
    CHK_INIT_IF_RETURN();

    for (uint8_t target_layer_index = layer_index + 1; target_layer_index < circuit.num_layers + 1;
         target_layer_index++) {
      auto [num_blocks, num_threads] =
        find_thread_block(1 << circuit.on_host_subset_num_vars[target_layer_index][layer_index]);

      extract_subset_evaluation<<<num_blocks, num_threads>>>(
        circuit.reverse_exts, layer_index, target_layer_index, evaluations, subset_evaluations);
    }

    return CHK_LAST();
  }
} // namespace virgo
