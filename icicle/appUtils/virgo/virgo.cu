#include "virgo.cuh"
#include "sumcheck.cu"
#include "merkle_tree.cu"
#include "circuit.cu"

namespace virgo {
  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkSumAllCase1)(
    const SumcheckConfig& config,
    curve_config::scalar_t* arr1,
    curve_config::scalar_t* arr2,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_sum_all_case_1<curve_config::scalar_t>(config, arr1, arr2, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkSumAllCase2)(
    const SumcheckConfig& config, curve_config::scalar_t* arr, curve_config::scalar_t* output, int n)
  {
    return bk_sum_all_case_2<curve_config::scalar_t>(config, arr, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkProduceCase1)(
    const SumcheckConfig& config,
    curve_config::scalar_t* table1,
    curve_config::scalar_t* table2,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_produce_case_1<curve_config::scalar_t>(config, table1, table2, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkProduceCase2)(
    const SumcheckConfig& config, curve_config::scalar_t* table, curve_config::scalar_t* output, int n)
  {
    return bk_produce_case_2<curve_config::scalar_t>(config, table, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkReduce)(
    const SumcheckConfig& config, curve_config::scalar_t* arr, int n, curve_config::scalar_t r)
  {
    return bk_reduce<curve_config::scalar_t>(config, arr, n, r);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BuildMerkleTree)(
    const MerkleTreeConfig<curve_config::scalar_t>& config, curve_config::scalar_t* tree, int n)
  {
    return build_merkle_tree<curve_config::scalar_t>(config, tree, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, HashMerkleTreeSlice)(
    const MerkleTreeConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* input,
    curve_config::scalar_t* output,
    int n,
    int slice_size)
  {
    return hash_merkle_tree_slice<curve_config::scalar_t>(config, input, output, n, slice_size);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, CircuitEvaluate)(
    const virgo::Circuit<curve_config::scalar_t>& circuit, curve_config::scalar_t** evaluations)
  {
    return circuit_evaluate<curve_config::scalar_t>(circuit, evaluations);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, CircuitSubsetEvaluations)(
    const virgo::Circuit<curve_config::scalar_t>& circuit,
    uint8_t layer_index,
    curve_config::scalar_t** evaluations,
    curve_config::scalar_t** subset_evaluations)
  {
    return circuit_subset_evaluations<curve_config::scalar_t>(circuit, layer_index, evaluations, subset_evaluations);
  }
} // namespace virgo
