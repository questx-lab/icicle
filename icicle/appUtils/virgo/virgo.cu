#include "hip/hip_runtime.h"
#include "virgo.cuh"
#include <thrust/device_vector.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/execution_policy.h>
#include <iostream>
#include <string>
#include <sstream>

namespace virgo {
  template <typename S>
    __global__ void mul_pair_kernel(S* arr1, S* arr2, S* result, S inv_r_mont2, int n) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    result[tid] = arr1[tid] * arr2[tid] * inv_r_mont2;
  }

  template <typename S>
  __global__ void reduce_sum_kernel(S* result, uint32_t n, uint32_t half) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    auto other = tid + half;
    if (other < n) {
      result[tid] = result[tid] + result[other];
    }
  }

    template <typename S>
  __global__ void reduce_sum_kernel2(S* result, uint32_t m, uint32_t n, uint32_t half, uint32_t offset) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    uint32_t arr_id = tid % m;
    uint32_t arr_pos = tid / m;

    auto other = arr_pos + half;
    if (other < n) {
      auto start = offset * arr_id + arr_pos;
      result[start] = result[start] + result[start + half];
    }
  }

  template <typename S>
  hipError_t sum_arrays(S* arrs, uint32_t m, uint32_t n)
  {
    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);
    hipStream_t stream;
    hipStreamCreate(&stream);

    auto x = n;
    while (x > 1) {
      int worker_count = (x * m + 1) >> 1;
      int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
      int num_blocks = (worker_count + num_threads - 1) / num_threads;

      int half = (x + 1) >> 1;
      reduce_sum_kernel2 <<< num_blocks, num_threads, 0, stream >>> (arrs, m, x, half, n);

      x = (x + 1) >> 1;
    }
  }

  template <typename S>
  hipError_t sum_single_array(S* arr, int n)
  {
    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);
    hipStream_t stream;
    hipStreamCreate(&stream);

    auto x = n;
    while (x > 1) {
      int worker_count = x >> 1;
      int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
      int num_blocks = (worker_count + num_threads - 1) / num_threads;

      int half = (x + 1) >> 1;
      reduce_sum_kernel <<< num_blocks, num_threads, 0, stream >>> (arr, x, half);

      x = (x + 1) >> 1;
    }
  }

  template <typename S>
  hipError_t bk_sum_all_case1(
    S* arr1, S* arr2, S* output, int n)
  {
    CHK_INIT_IF_RETURN();

    int cuda_device_ix = 0;
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, cuda_device_ix);

    int worker_count = n;
    int num_threads = worker_count < prop.maxThreadsPerBlock ? worker_count : prop.maxThreadsPerBlock;
    int num_blocks = (worker_count + num_threads - 1) / num_threads;

    const int log_n = log2(n);
    hipStream_t stream;
    hipStreamCreate(&stream);

    S* device_tmp;
    // allocate device array
    hipMalloc((void**)&device_tmp, n * sizeof(S));

    // This is arkwork inverse R, not icicle inverse R.
    // inv_r = 9915499612839321149637521777990102151350674507940716049588462388200839649614
    // S inv_r_mont({0x6db1194e, 0xdc5ba005, 0xe111ec87, 0x90ef5a9, 0xaeb85d5d, 0xc8260de4, 0x82c5551c, 0x15ebf951});
    // inv_r2 = inv_r ^ 2 = 8519677608991584271437967308266649112183478179623991153221810821821888926024
    S inv_r_mont2({0xd3c71148, 0xae12ba81, 0xb38e2428, 0x52f28270, 0x79a1edeb, 0xe065f3e3, 0xe436631e, 0x12d5f775});

    mul_pair_kernel <<< num_blocks, num_threads, 0, stream >>> (arr1, arr2, device_tmp, inv_r_mont2, n);

    sum_single_array(device_tmp, n);
    hipMemcpy(output, device_tmp, sizeof(S), hipMemcpyDeviceToHost);

    // S* tmp2;
    // hipMalloc((void**)&tmp2, 2 * n * sizeof(S));
    // hipMemcpy(tmp2, device_tmp, n * sizeof(S), hipMemcpyHostToHost);
    // hipMemcpy(tmp2 + n, device_tmp, n * sizeof(S), hipMemcpyHostToHost);

    // sum_arrays(tmp2, 2, n);

    // hipMemcpy(output, tmp2, sizeof(S), hipMemcpyDeviceToHost);
    // std::cout << "output 0 = " << output << std::endl;

    // hipMemcpy(output, tmp2 + n, sizeof(S), hipMemcpyDeviceToHost);
    // std::cout << "output 1 = " << output << std::endl;

    hipFree(device_tmp);

    return CHK_LAST();
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkSumAllCase1)(
    curve_config::scalar_t* arr1,
    curve_config::scalar_t* arr2,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_sum_all_case1<curve_config::scalar_t>(arr1, arr2, output, n);
  }
}
