#include "virgo.cuh"
#include "sumcheck.cu"
#include "merkle_tree.cu"
#include "circuit.cu"
#include "gkr.cu"

namespace virgo {
  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkSumAllCase1)(
    const SumcheckConfig& config,
    curve_config::scalar_t* arr1,
    curve_config::scalar_t* arr2,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_sum_all_case_1<curve_config::scalar_t>(config, arr1, arr2, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkSumAllCase2)(
    const SumcheckConfig& config, curve_config::scalar_t* arr, curve_config::scalar_t* output, int n)
  {
    return bk_sum_all_case_2<curve_config::scalar_t>(config, arr, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkProduceCase1)(
    const SumcheckConfig& config,
    curve_config::scalar_t* table1,
    curve_config::scalar_t* table2,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_produce_case_1<curve_config::scalar_t>(config, table1, table2, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkProduceCase2)(
    const SumcheckConfig& config, curve_config::scalar_t* table, curve_config::scalar_t* output, int n)
  {
    return bk_produce_case_2<curve_config::scalar_t>(config, table, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkReduce)(
    const SumcheckConfig& config, curve_config::scalar_t* arr, int n, curve_config::scalar_t r)
  {
    return bk_reduce<curve_config::scalar_t>(config, arr, n, r);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BuildMerkleTree)(
    const MerkleTreeConfig<curve_config::scalar_t>& config, curve_config::scalar_t* tree, int n)
  {
    return build_merkle_tree<curve_config::scalar_t>(config, tree, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, HashMerkleTreeSlice)(
    const MerkleTreeConfig<curve_config::scalar_t>& config,
    curve_config::scalar_t* input,
    curve_config::scalar_t* output,
    int n,
    int slice_size)
  {
    return hash_merkle_tree_slice<curve_config::scalar_t>(config, input, output, n, slice_size);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, CircuitEvaluate)(
    const virgo::Circuit<curve_config::scalar_t>& circuit,
    uint32_t num_subcircuits,
    curve_config::scalar_t** evaluations)
  {
    return circuit_evaluate<curve_config::scalar_t>(circuit, num_subcircuits, evaluations);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, CircuitSubsetEvaluations)(
    const virgo::Circuit<curve_config::scalar_t>& circuit,
    uint32_t num_subcircuits,
    uint8_t layer_index,
    curve_config::scalar_t** evaluations,
    curve_config::scalar_t** subset_evaluations)
  {
    return circuit_subset_evaluations<curve_config::scalar_t>(
      circuit, num_subcircuits, layer_index, evaluations, subset_evaluations);
  }

  extern "C" hipError_t
  CONCAT_EXPAND(CURVE, MulByScalar)(curve_config::scalar_t* arr, curve_config::scalar_t scalar, uint32_t n)
  {
    return mul_by_scalar(arr, scalar, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, PrecomputeBookeeping)(
    curve_config::scalar_t init, curve_config::scalar_t* g, uint8_t g_size, curve_config::scalar_t* output)
  {
    return precompute_bookeeping(init, g, g_size, output);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, InitializePhase1Plus)(
    uint32_t num_replicas,
    uint32_t num_layers,
    uint32_t output_size,
    SparseMultilinearExtension<curve_config::scalar_t>* f_extensions,
    curve_config::scalar_t** s_evaluations,
    curve_config::scalar_t* bookeeping_g,
    curve_config::scalar_t* output)
  {
    return initialize_phase_1_plus(
      num_replicas, num_layers, output_size, f_extensions, s_evaluations, bookeeping_g, output);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, InitializePhase2Plus)(
    uint32_t num_replicas,
    uint32_t num_layers,
    uint32_t* on_host_output_size,
    SparseMultilinearExtension<curve_config::scalar_t>* f_extensions,
    curve_config::scalar_t* bookeeping_g,
    curve_config::scalar_t* bookeeping_u,
    curve_config::scalar_t** output)
  {
    return initialize_phase_2_plus(
      num_replicas, num_layers, on_host_output_size, f_extensions, bookeeping_g, bookeeping_u, output);
  }
} // namespace virgo
