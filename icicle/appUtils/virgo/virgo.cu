#include "virgo.cuh"
#include "sumcheck.cu"
#include "merkle_tree.cu"

namespace virgo {
  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkSumAllCase1)(
    const SumcheckConfig &config,
    curve_config::scalar_t* arr1,
    curve_config::scalar_t* arr2,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_sum_all_case_1<curve_config::scalar_t>(config, arr1, arr2, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkSumAllCase2)(
    const SumcheckConfig &config,
    curve_config::scalar_t* arr,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_sum_all_case_2<curve_config::scalar_t>(config, arr, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkProduceCase1)(
    const SumcheckConfig &config,
    curve_config::scalar_t* table1,
    curve_config::scalar_t* table2,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_produce_case_1<curve_config::scalar_t>(config, table1, table2, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BkProduceCase2)(
    const SumcheckConfig &config,
    curve_config::scalar_t* table,
    curve_config::scalar_t* output,
    int n)
  {
    return bk_produce_case_2<curve_config::scalar_t>(config, table, output, n);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, BuildMerkleTree) (
    const MerkleTreeConfig<curve_config::scalar_t> &config,
    curve_config::scalar_t* tree,
    int n,
    int slice_size)
  {
    return build_merkle_tree<curve_config::scalar_t>(config, tree, n, slice_size);
  }

  extern "C" hipError_t CONCAT_EXPAND(CURVE, HashMerkleTreeSlice) (
    const MerkleTreeConfig<curve_config::scalar_t> &config,
    curve_config::scalar_t* input,
    curve_config::scalar_t* output,
    int n,
    int slice_size)
  {
    return hash_merkle_tree_slice<curve_config::scalar_t>(config, input, output, n, slice_size);
  }
}
