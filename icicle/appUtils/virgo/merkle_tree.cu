#include "hip/hip_runtime.h"
#include "virgo.cuh"
#include "common.cuh"
#include <iostream>
#include <string>
#include <chrono>
#include <sstream>
#include <tuple>

namespace virgo {
  template <typename S>
  __device__ S hash_one_field(S x) {
    auto result = x * x;
    result = result * x;

    return result * inv_r_mont2<S>;
  }

  /**
   * Runs only on device to hash an array of numbers and store the output in an output array.
   */
  template <typename S>
  __device__ void device_mimc_hash_array(uint32_t tid, const MerkleTreeConfig<S> config, S* arr,
      S* output, int n) {
    auto num_repetitions = config.max_mimc_k / n;
    if (num_repetitions < 2) {
      num_repetitions = 2;
    }

    auto r = S::from(0);
    for (int repetition_index = 0; repetition_index < num_repetitions; repetition_index++) {
      uint32_t d = config.D[repetition_index % 8];

      auto start = repetition_index % n;
      for (int i = 0; i < n; i++) {
        auto k_index = (repetition_index * n + i) % config.max_mimc_k;
        auto v_index = (start + d * i) % n;

        r = hash_one_field(r + arr[v_index] + config.mimc_params[k_index]);
      }
    }

    output[tid] = r;
  }

  template <typename S>
  __global__ void mimc_hash_array(const MerkleTreeConfig<S> config, S* arr, S* output, int n) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    device_mimc_hash_array(tid, config, arr + (tid << 1), output, n);
  }

  template <typename S>
  hipError_t build_merkle_tree_no_slice(const MerkleTreeConfig<S>& config, S* tree, int n) {
    auto stream = config.ctx.stream;

    auto x = n;
    auto offset = 0;
    while (x > 1) {
      auto [num_blocks, num_threads] = find_thread_block(x >> 1);
      mimc_hash_array <<< num_blocks, num_threads, 0, stream >>> (config, tree + offset, tree + offset + x, 2);
      offset += x;
      x = x / 2;
    }

    return CHK_LAST();
  }

  template <typename S>
  __global__ void hash_slice(const MerkleTreeConfig<S> config, S* arr, S* output, int n, int slice_size) {
    uint32_t tid = threadIdx.x + blockIdx.x * blockDim.x;

    device_mimc_hash_array(0, config, arr + tid * slice_size, output + tid, slice_size);
  }

  template <typename S>
  hipError_t hash_merkle_tree_slice(const MerkleTreeConfig<S>& config, S* input, S* output, int n, int slice_size) {
    auto slice_count = n / slice_size;
    auto [num_blocks, num_threads] = find_thread_block(slice_count);
    hash_slice <<< num_blocks, num_threads >>> (config, input, output, n, slice_size);

    return CHK_LAST();
  }

  template <typename S>
  hipError_t build_merkle_tree(const MerkleTreeConfig<S>& config, S* tree, int n, int slice_size) {
    return build_merkle_tree_no_slice(config, tree, n);
  }
}
